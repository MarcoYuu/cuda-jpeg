#include "hip/hip_runtime.h"
/*
 * cuda_main.cpp
 *
 *  Created on: 2012/11/12
 *      Author: momma
 */

#include <fstream>
#include <iostream>
#include <sstream>
#include <cmath>

#include <boost/lexical_cast.hpp>

#include "jpeg/cpu/cpu_jpeg.h"
#include "jpeg/cuda/cuda_jpeg.cuh"

#include "utils/debug_log.h"
#include "utils/util_cv.h"
#include "utils/type_definitions.h"

#include "utils/cuda/cuda_timer.h"
#include "utils/cuda/cuda_memory.hpp"

using namespace std;
using namespace util;
using namespace util::cuda;
using namespace jpeg;
using namespace jpeg::cuda;

struct TableExport: public util::DebugLog::OutputFormat {
private:
	const size_t size_;
	jpeg::cuda::CudaTable &table_;

public:
	TableExport(jpeg::cuda::CudaTable &yuv, int block_size) :
		size_(block_size),
		table_(yuv) {
	}
	void operator()(std::ofstream& ofs) const {
		table_.sync_to_host();
		ofs << table_.size() << std::endl;
		for (int i = 0; i < table_.size(); ++i) {
			if (i % size_ == 0)
				ofs << "\n\n\nblock :" << i / size_;

			if (i % 8 == 0)
				ofs << std::endl;

			if (i % 64 == 0)
				ofs << "\n8x8block :" << i << std::endl;

			//ofs << "(" << i << " -> " << table_[i].y << ", " << table_[i].u << ", " << table_[i].v << "), ";
			ofs << "(" << i << ", " << table_[i].y << "), ";
		}
	}
};

/**
 * @brief ログファイル出力用
 *
 * @author yuumomma
 * @version 1.0
 */
template<class CudaMemory>
struct BlockExport: public util::DebugLog::OutputFormat {
private:
	const size_t size_;
	CudaMemory &yuv_;

public:
	BlockExport(CudaMemory &yuv, int block_size) :
		size_(block_size),
		yuv_(yuv) {

	}
	void operator()(std::ofstream& ofs) const {
		yuv_.sync_to_host();
		ofs << yuv_.size() << std::endl;
		for (int i = 0; i < yuv_.size(); ++i) {
			if (i % size_ == 0)
				ofs << "\n\n\nblock :" << i / size_;

			if (i % 8 == 0)
				ofs << std::endl;

			if (i % 64 == 0)
				ofs << "\n8x8block :" << i << std::endl;

			ofs << (int) (yuv_[i]) << ", ";
		}
	}
};

void CalcurateMatrixTest();

void encoder_decoder(const std::string& file_name, const std::string& out_file_name, size_t block_width,
	size_t block_height, int quarity);

void encode_and_decode(const std::string& file_name, const std::string& out_file_name, size_t block_width,
	size_t block_height, int quarity);

void color_conversion_only(const std::string &file_name, const std::string &out_file_name, size_t block_width,
	size_t block_height, int quarity);

void code_func(const std::string &file_name, const std::string &out_file_name, size_t block_width,
	size_t block_height, int quarity);

void cuda_main(const std::string &file_name, const std::string &out_file_name, size_t block_width,
	size_t block_height, int quarity) {
	code_func(file_name, out_file_name, block_width, block_height, quarity);
	//encoder_decoder(file_name, out_file_name, block_width, block_height, quarity);
	//encode_and_decode(file_name, out_file_name, block_width, block_height, quarity);
	//color_conversion_only(file_name, out_file_name, block_width, block_height, quarity);
}

void encode(const byte* rgb, byte* huffman, size_t width, size_t height, size_t block_width,
	size_t block_height, int quarity) {

	// ブロックサイズ
	const int IMG_MEM_SIZE = width * height * 3 / 2;
	const int BLOCK_WIDTH = block_width == 0 ? width : block_width;
	const int BLOCK_HEIGHT = block_height == 0 ? height : block_height;
	const int BLOCK_MEM_SIZE = BLOCK_WIDTH * BLOCK_HEIGHT * 3 / 2;
	const int BLOCK_NUM = width * height / (BLOCK_WIDTH * BLOCK_HEIGHT);

	DeviceByteBuffer encode_src(rgb, width * height * 3);
	CudaTable encode_table(width * height);
	CudaByteBuffer encode_yuv_result(IMG_MEM_SIZE);
	CudaIntBuffer encode_dct_result(IMG_MEM_SIZE);
	CudaIntBuffer encode_qua_result(IMG_MEM_SIZE);
	CudaByteBuffer encode_huffman_result(IMG_MEM_SIZE);
	IntBuffer huffman_effective_bits(BLOCK_NUM);

	CreateConversionTable(width, height, BLOCK_WIDTH, BLOCK_HEIGHT, encode_table);
	ConvertRGBToYUV(encode_src, encode_yuv_result, width, height, BLOCK_WIDTH, BLOCK_HEIGHT, encode_table);
	DiscreteCosineTransform(encode_yuv_result, encode_dct_result);
	ZigzagQuantize(encode_dct_result, encode_qua_result, BLOCK_MEM_SIZE, quarity);

	encode_huffman_result.fill_zero();
	HuffmanEncode(encode_qua_result, encode_huffman_result, huffman_effective_bits);
	encode_huffman_result.copy_to_host(huffman, IMG_MEM_SIZE);
}

void decode(const byte *huffman, byte *dst, size_t width, size_t height, size_t block_width,
	size_t block_height, int quarity) {

	// ブロックサイズ
	const int IMG_MEM_SIZE = width * height * 3 / 2;
	const int BLOCK_WIDTH = block_width == 0 ? width : block_width;
	const int BLOCK_HEIGHT = block_height == 0 ? height : block_height;
	const int BLOCK_MEM_SIZE = BLOCK_WIDTH * BLOCK_HEIGHT * 3 / 2;

	CudaTable decode_table(BLOCK_WIDTH * BLOCK_HEIGHT);
	CudaByteBuffer decode_huffman_src(BLOCK_MEM_SIZE);
	CudaIntBuffer decode_qua_src(BLOCK_MEM_SIZE);
	CudaIntBuffer decode_dct_src(BLOCK_MEM_SIZE);
	CudaByteBuffer decode_yuv_src(BLOCK_MEM_SIZE);
	CudaByteBuffer decode_result(BLOCK_WIDTH * BLOCK_HEIGHT * 3);

	CreateConversionTable(BLOCK_WIDTH, BLOCK_HEIGHT, BLOCK_WIDTH, BLOCK_HEIGHT, decode_table);

	InBitStream ibs(huffman, IMG_MEM_SIZE);
	cpu::decode_huffman(&ibs, decode_qua_src.host_data(), BLOCK_WIDTH, BLOCK_HEIGHT);
	decode_qua_src.sync_to_device();

	InverseZigzagQuantize(decode_qua_src, decode_dct_src, BLOCK_MEM_SIZE, quarity);

	InverseDiscreteCosineTransform(decode_dct_src, decode_yuv_src);
	decode_yuv_src.sync_to_host();

	ConvertYUVToRGB(decode_yuv_src, decode_result, BLOCK_WIDTH, BLOCK_HEIGHT, BLOCK_WIDTH, BLOCK_HEIGHT,
		decode_table);
	decode_result.copy_to_host(dst, decode_result.size());
}

void code_func(const std::string &file_name, const std::string &out_file_name, size_t block_width,
	size_t block_height, int quarity) {

	// 画像読み込み
	BitmapCVUtil source(file_name, BitmapCVUtil::RGB_COLOR);
	const int width = source.getWidth();
	const int height = source.getHeight();

	ByteBuffer huffman(width * height * 3 / 2);
	encode((byte*) source.getRawData(), huffman.data(), width, height, block_width, block_height, quarity);

	const int BLOCK_NUM = width * height / (block_width * block_height);
	BitmapCVUtil bmp(block_width, block_height, 8, source.getBytePerPixel());
	for (int i = 0; i < BLOCK_NUM; ++i) {
		decode(huffman.data() + huffman.size() / BLOCK_NUM * i, (byte*) bmp.getRawData(), block_width,
			block_height, block_width, block_height, quarity);

		string index = boost::lexical_cast<string>(i);
		string qrty = boost::lexical_cast<string>(quarity);
		string outname = "cuda_" + index + "_" + qrty + "_" + out_file_name;
		DebugLog::log("export to file :" + outname);
		bmp.saveToFile(outname);
	}
}

void encoder_decoder(const std::string& file_name, const std::string& out_file_name, size_t block_width,
	size_t block_height, int quarity) {

	using namespace std;
	using namespace util;
	using namespace util::cuda;
	using namespace jpeg;
	using namespace jpeg::cuda;

	DebugLog::initTimer(true);

	// 画像読み込み
	BitmapCVUtil source(file_name, BitmapCVUtil::RGB_COLOR);
	const int width = source.getWidth();
	const int height = source.getHeight();
	const int block_num = width * height / (block_width * block_height);

	DebugLog::startSection("Start CUDA Encoding & Decoding");

	DeviceByteBuffer encode_src(width * height * 3);
	CudaByteBuffer encode_result(width * height * 3 / 2);
	IntBuffer effective_bits(block_num);
	Encoder encoder(width, height, block_width, block_height);
	encoder.setQuarity(quarity);
	{
		DebugLog::startLoggingTime("Encode");

		encode_src.write_device((byte*) ((source.getRawData())), encode_src.size());
		encoder.encode(encode_src, encode_result, effective_bits);
		encode_result.sync_to_host();

		DebugLog::endLoggingTime();
	}

	BitmapCVUtil bmp(block_width, block_height, 8, source.getBytePerPixel());
	CudaByteBuffer decode_result(block_width * block_height * 3);
	Decoder decoder(block_width, block_height);
	decoder.setQuarity(quarity);
	{
		for (int i = 0; i < block_num; ++i) {
			DebugLog::startLoggingTime("Decode");

			decoder.decode(encode_result.host_data(), encode_result.size() / block_num, decode_result);
			decode_result.sync_to_host();

			DebugLog::endLoggingTime();

			string index = boost::lexical_cast<string>(i);
			string qrty = boost::lexical_cast<string>(quarity);
			string outname = "cuda_" + index + "_" + qrty + "_" + out_file_name;
			decode_result.copy_to_host((byte*) ((bmp.getRawData())), decode_result.size());
			bmp.saveToFile(outname);
			DebugLog::log("export to file :" + outname);
		}
	}

	DebugLog::endSection("Finish CUDA Encoding & Decoding");
}

void encode_and_decode(const std::string& file_name, const std::string& out_file_name, size_t block_width,
	size_t block_height, int quarity) {

	DebugLog::initTimer(true);

	// 画像読み込み
	BitmapCVUtil source(file_name, BitmapCVUtil::RGB_COLOR);
	const int width = source.getWidth();
	const int height = source.getHeight();

	// ブロックサイズ
	const int IMG_MEM_SIZE = width * height * 3 / 2;
	const int BLOCK_WIDTH = block_width == 0 ? width : block_width;
	const int BLOCK_HEIGHT = block_height == 0 ? height : block_height;
	const int BLOCK_MEM_SIZE = BLOCK_WIDTH * BLOCK_HEIGHT * 3 / 2;
	const int BLOCK_NUM = width * height / (BLOCK_WIDTH * BLOCK_HEIGHT);

	DebugLog::startSection("Start CUDA Encoding & Decoding");

	DebugLog::log("infile_name: " + file_name);
	DebugLog::log("outfile_name: " + out_file_name);
	DebugLog::log("block_width: " + boost::lexical_cast<string>(block_width));
	DebugLog::log("block_height: " + boost::lexical_cast<string>(block_height));
	DebugLog::log("quarity: " + boost::lexical_cast<string>(quarity));

	DeviceByteBuffer encode_src((byte*) ((source.getRawData())), width * height * 3);
	CudaTable encode_table(width * height);
	CudaByteBuffer encode_yuv_result(IMG_MEM_SIZE);
	CudaIntBuffer encode_dct_result(IMG_MEM_SIZE);
	CudaIntBuffer encode_qua_result(IMG_MEM_SIZE);
	CudaByteBuffer encode_huffman_result(IMG_MEM_SIZE);
	IntBuffer huffman_effective_bits(BLOCK_NUM);
	{
		DebugLog::startSubSection("Encode");
		DebugLog::startLoggingTime("CreateConvertTable");
		CreateConversionTable(width, height, BLOCK_WIDTH, BLOCK_HEIGHT, encode_table);
		DebugLog::endLoggingTime();

		DebugLog::startLoggingTime("ConvertRGBToYUV");
		ConvertRGBToYUV(encode_src, encode_yuv_result, width, height, BLOCK_WIDTH, BLOCK_HEIGHT,
			encode_table);
		DebugLog::endLoggingTime();
		DebugLog::exportToFile("encode_yuv_result.txt",
			BlockExport<CudaByteBuffer>(encode_yuv_result, BLOCK_MEM_SIZE));

		DebugLog::startLoggingTime("DiscreteCosineTransform");
		DiscreteCosineTransform(encode_yuv_result, encode_dct_result);
		DebugLog::endLoggingTime();
		DebugLog::exportToFile("encode_dct_result.txt",
			BlockExport<CudaIntBuffer>(encode_dct_result, BLOCK_MEM_SIZE));

		DebugLog::startLoggingTime("ZigzagQuantize");
		ZigzagQuantize(encode_dct_result, encode_qua_result, BLOCK_MEM_SIZE, quarity);
		DebugLog::endLoggingTime();
		DebugLog::exportToFile("encode_qua_result.txt",
			BlockExport<CudaIntBuffer>(encode_qua_result, BLOCK_MEM_SIZE));

		DebugLog::startLoggingTime("HuffmanEncode");
		encode_huffman_result.fill_zero();
		HuffmanEncode(encode_qua_result, encode_huffman_result, huffman_effective_bits);
		encode_huffman_result.sync_to_host();
		DebugLog::endLoggingTime();

		DebugLog::dump_memory(encode_huffman_result.host_data(), huffman_effective_bits[0] / 8 + 1,
			"huffman.finish.dat");

		DebugLog::printTotalTime();
		DebugLog::endSubSection();
	}

	DebugLog::resetTotalTime();
	BitmapCVUtil bmp(BLOCK_WIDTH, BLOCK_HEIGHT, 8, source.getBytePerPixel());

	CudaTable decode_table(BLOCK_WIDTH * BLOCK_HEIGHT);
	CudaByteBuffer decode_huffman_src(BLOCK_MEM_SIZE);
	CudaIntBuffer decode_qua_src(BLOCK_MEM_SIZE);
	CudaIntBuffer decode_dct_src(BLOCK_MEM_SIZE);
	CudaByteBuffer decode_yuv_src(BLOCK_MEM_SIZE);
	CudaByteBuffer decode_result(BLOCK_WIDTH * BLOCK_HEIGHT * 3);
	{
		DebugLog::startSubSection("Decode");

		DebugLog::startLoggingTime("CreateConvertTable");
		CreateConversionTable(BLOCK_WIDTH, BLOCK_HEIGHT, BLOCK_WIDTH, BLOCK_HEIGHT, decode_table);
		DebugLog::endLoggingTime();

		for (int i = 0; i < width / BLOCK_WIDTH * height / BLOCK_HEIGHT; ++i) {
			string index = boost::lexical_cast<string>(i);

			InBitStream ibs(encode_huffman_result.host_data() + encode_huffman_result.size() / BLOCK_NUM * i,
				encode_huffman_result.size() / BLOCK_NUM);
			cpu::decode_huffman(&ibs, decode_qua_src.host_data(), BLOCK_WIDTH, BLOCK_HEIGHT);
			decode_qua_src.sync_to_device();
			DebugLog::exportToFile("decode_qua_src" + index + ".txt",
				BlockExport<CudaIntBuffer>(decode_qua_src, BLOCK_MEM_SIZE));

			DebugLog::startLoggingTime("InverseZigzagQuantize");
			InverseZigzagQuantize(decode_qua_src, decode_dct_src, BLOCK_MEM_SIZE, quarity);
			DebugLog::endLoggingTime();
			DebugLog::exportToFile("decode_dct_src" + index + ".txt",
				BlockExport<CudaIntBuffer>(decode_dct_src, BLOCK_MEM_SIZE));

			DebugLog::startLoggingTime("InverseDiscreteCosineTransform");
			InverseDiscreteCosineTransform(decode_dct_src, decode_yuv_src);
			DebugLog::endLoggingTime();
			DebugLog::exportToFile("decode_yuv_src" + index + ".txt",
				BlockExport<CudaByteBuffer>(decode_yuv_src, BLOCK_MEM_SIZE));
			decode_yuv_src.sync_to_host();

			DebugLog::startLoggingTime("ConvertYUVToRGB");
			ConvertYUVToRGB(decode_yuv_src, decode_result, BLOCK_WIDTH, BLOCK_HEIGHT, BLOCK_WIDTH,
				BLOCK_HEIGHT, decode_table);
			decode_result.copy_to_host((byte*) ((bmp.getRawData())), decode_result.size());
			DebugLog::endLoggingTime();

			string qrty = boost::lexical_cast<string>(quarity);
			string outname = "cuda_" + index + "_" + qrty + "_" + out_file_name;
			DebugLog::log("export to file :" + outname);
			bmp.saveToFile(outname);
		}
		DebugLog::printTotalTime();
		DebugLog::endSubSection();
	}
	DebugLog::endSection("Finish CUDA Encoding & Decoding");
}

void color_conversion_only(const std::string &file_name, const std::string &out_file_name, size_t block_width,
	size_t block_height, int quarity) {

	// 画像読み込み
	BitmapCVUtil source(file_name, BitmapCVUtil::RGB_COLOR);
	const int width = source.getWidth();
	const int height = source.getHeight();

	// ブロックサイズ
	const int BLOCK_WIDTH = block_width == 0 ? width : block_width;
	const int BLOCK_HEIGHT = block_height == 0 ? height : block_height;
	const int BLOCK_MEM_SIZE = BLOCK_WIDTH * BLOCK_HEIGHT * 3 / 2;
	const int IMG_MEM_SIZE = width * height * 3 / 2;

	DebugLog::startSection("Start CUDA Encoding & Decoding");

	DeviceByteBuffer encode_src((byte*) (source.getRawData()), width * height * 3);
	CudaTable encode_table(width * height);
	CudaByteBuffer encode_yuv_result(IMG_MEM_SIZE);
	{
		DebugLog::startSubSection("Encode");

		CreateConversionTable(width, height, BLOCK_WIDTH, BLOCK_HEIGHT, encode_table);
		DebugLog::exportToFile("encode_table.txt", TableExport(encode_table, BLOCK_WIDTH * BLOCK_HEIGHT));

		ConvertRGBToYUV(encode_src, encode_yuv_result, width, height, BLOCK_WIDTH, BLOCK_HEIGHT,
			encode_table);
		encode_yuv_result.sync_to_host();
		DebugLog::exportToFile("encode_yuv_result.txt",
			BlockExport<CudaByteBuffer>(encode_yuv_result, BLOCK_MEM_SIZE));

		DebugLog::endSubSection();
	}

	BitmapCVUtil bmp(BLOCK_WIDTH, BLOCK_HEIGHT, 8, source.getBytePerPixel());
	CudaTable decode_table(BLOCK_WIDTH * BLOCK_HEIGHT);
	CudaByteBuffer decode_yuv_src(BLOCK_MEM_SIZE);
	CudaByteBuffer decode_result(BLOCK_WIDTH * BLOCK_HEIGHT * 3);
	{
		DebugLog::startSubSection("Decode");

		CreateConversionTable(BLOCK_WIDTH, BLOCK_HEIGHT, BLOCK_WIDTH, BLOCK_HEIGHT, decode_table);
		DebugLog::exportToFile("decode_table.txt", TableExport(decode_table, BLOCK_WIDTH * BLOCK_HEIGHT));

		for (int i = 0; i < width / BLOCK_WIDTH * height / BLOCK_HEIGHT; ++i) {
			string index = boost::lexical_cast<string>(i);

			DebugLog::log("copy block memory.");
			decode_yuv_src.write_device(encode_yuv_result.host_data() + i * decode_yuv_src.size(),
				decode_yuv_src.size());

			ConvertYUVToRGB(decode_yuv_src, decode_result, BLOCK_WIDTH, BLOCK_HEIGHT, BLOCK_WIDTH,
				BLOCK_HEIGHT, decode_table);
			decode_result.copy_to_host((byte*) (bmp.getRawData()), decode_result.size());
			DebugLog::exportToFile("decode_yuv_src" + index + ".txt",
				BlockExport<CudaByteBuffer>(decode_yuv_src, BLOCK_MEM_SIZE));
			decode_yuv_src.sync_to_host();

			string outname = "cudacolor_" + index + "_" + boost::lexical_cast<string>(quarity) + "_"
				+ out_file_name;

			DebugLog::log("export to file :" + outname);

			bmp.saveToFile(outname);
		}

		DebugLog::endSubSection();
	}

	DebugLog::endSection("");
}

void CalcurateMatrixTest() {
	using namespace jpeg::cuda;

	float DCT[64];
	CalculateDCTMatrix(DCT);
	float iDCT[64];
	CalculateiDCTMatrix(iDCT);
	for (int i = 0; i < 64; ++i) {
		if (i % 8 == 0)
			printf("\n");

		printf("%12.8f, ", DCT[i]);
	}
	std::cout << std::endl;
	for (int i = 0; i < 64; ++i) {
		if (i % 8 == 0)
			printf("\n");

		printf("%12.8f, ", iDCT[i]);
	}
}

