#include <cstdlib>
#include <cstring>
#include <cmath>

#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>

#include "cpu_jpeg.h"
#include "gpu_jpeg.cuh"

#include "utils/util_cv.h"
#include "utils/timer.h"
#include "utils/cuda_timer.h"
#include "utils/cuda_memory.hpp"
#include "utils/util_cv.h"
#include "utils/encoder_tables.h"
#include "utils/in_bit_stream.h"
#include "utils/gpu_in_bit_stream.cuh"

using namespace std;

void gpu_exec(const std::string &file_name, const std::string &out_file_name) {
	//----------------------------------------------------------------------------
	// 画像読み込み
	//============================================================================
	BitmapCVUtil source(file_name, BitmapCVUtil::RGB_COLOR);
	const int width = source.getWidth();
	const int height = source.getHeight();
	const int Y_size = width * height;
	const int C_size = Y_size / 4;
	const int YCC_size = Y_size + C_size * 2;

	std::cout << "===============================================" << std::endl;
	std::cout << " Start GPU Encoding & Decoding" << std::endl;
	std::cout << "-----------------------------------------------\n" << std::endl;

	//----------------------------------------------------------------------------
	// カーネルDimension設定
	//============================================================================
	const int THREADS = 256;
	const int DCT4_TH = 1;
	const int QUA0_TH = 64;
	const int QUA1_TH = 64;
	const int HUF0_TH = 16;
	const int HUF1_TH = 4; //divide使うなら最速

	dim3 Dg0_0(Y_size / THREADS, 1, 1), Db0_0(THREADS, 1, 1);
	dim3 Dg0_1(Y_size / THREADS / 2, 1, 1), Db0_1(height / 2, 1, 1);
	dim3 Dg1((YCC_size) / 64 / DCT4_TH, 1, 1), Db1(DCT4_TH, 8, 8); //DCT4_THは16が最大
	dim3 Dg2_0(Y_size / QUA0_TH, 1, 1), Db2_0(QUA0_TH, 1, 1);
	dim3 Dg2_1((2 * C_size) / QUA1_TH, 1, 1), Db2_1(QUA1_TH, 1, 1);
	dim3 Dg3_0(YCC_size / 64 / HUF0_TH, 1, 1), Db3_0(HUF0_TH, 1, 1); //YCC_size
	dim3 Dg3_1(YCC_size / 64 / HUF1_TH, 1, 1), Db3_1(HUF1_TH, 1, 1); //YCC_size

	//----------------------------------------------------------------------------
	// 処理開始
	//============================================================================
	CudaStopWatch watch;

	int result_size;
	ByteBuffer num_bits(YCC_size / 64);
	cuda_memory<byte> encode_result(sizeof(byte) * (Y_size * 3));
	encode_result.fillZero();
	{
		//----------------------------------------------------------------------------
		// 色変換テーブルの作成 Encode用定数転送
		//============================================================================
		cuda_memory<int> trans_table_Y(Y_size);
		cuda_memory<int> trans_table_C(Y_size);
		make_trans_table(trans_table_Y.host_data(), trans_table_C.host_data(), width, height);
		trans_table_C.sync_to_device();
		trans_table_Y.sync_to_device();

		//----------------------------------------------------------------------------
		// メモリ確保
		//============================================================================
		device_memory<byte> src(Y_size * 3);
		device_memory<int> yuv_buffer(YCC_size);
		device_memory<int> quantized(YCC_size);
		device_memory<int> dct_coeficient(YCC_size);
		device_memory<float> dct_tmp_buffer(YCC_size);

		GPUJpegOutBitStream out_bit_stream(YCC_size / 64, MBS);

		std::cout << "	-----------------------------------------------" << std::endl;
		std::cout << "	 Encode" << std::endl;
		std::cout << "	-----------------------------------------------" << std::endl;
		watch.start();
		{
			//----------------------------------------------------------------------------
			// メモリ転送
			//============================================================================
			src.write_device((byte*) source.getRawData(), width * height * 3);

			//----------------------------------------------------------------------------
			// RGB->yuv
			//============================================================================
			gpu_color_trans_Y<<<Dg0_0, Db0_0>>>(src.device_data(), yuv_buffer.device_data(), trans_table_Y.device_data());
			gpu_color_trans_C<<<Dg0_0, Db0_0>>>(src.device_data(), yuv_buffer.device_data(), trans_table_C.device_data(), height, C_size);

			//----------------------------------------------------------------------------
			// DCT
			//============================================================================
			gpu_dct_0<<<Dg1, Db1>>>(yuv_buffer.device_data(), dct_tmp_buffer.device_data());
			gpu_dct_1<<<Dg1, Db1>>>(dct_tmp_buffer.device_data(), dct_coeficient.device_data());

			//----------------------------------------------------------------------------
			// 量子化
			//============================================================================
			gpu_zig_quantize_Y<<<Dg2_0, Db2_0>>>(dct_coeficient.device_data(), quantized.device_data());
			gpu_zig_quantize_C<<<Dg2_1, Db2_1>>>(dct_coeficient.device_data(), quantized.device_data(), Y_size);
			//----------------------------------------------------------------------------
			// ハフマン符号化
			//============================================================================
			gpu_huffman_mcu<<<Dg3_0, Db3_0>>>(quantized.device_data(), out_bit_stream.status().device_data(),
				out_bit_stream.writable_head(), out_bit_stream.end(), width, height);

			// 逐次処理のためCPUに戻す
			out_bit_stream.status().sync_to_host();
			cpu_huffman_middle(out_bit_stream.status().host_data(), width, height, num_bits.data());
			out_bit_stream.status().sync_to_device();

			gpu_huffman_write_devide0<<<Dg3_1, Db3_1>>>(out_bit_stream.status().device_data(),
				out_bit_stream.writable_head(), encode_result.device_data(), width, height);
			gpu_huffman_write_devide1<<<Dg3_1, Db3_1>>>(out_bit_stream.status().device_data(),
				out_bit_stream.writable_head(), encode_result.device_data(), width, height);
			gpu_huffman_write_devide2<<<Dg3_1, Db3_1>>>(out_bit_stream.status().device_data(),
				out_bit_stream.writable_head(), encode_result.device_data(), width, height);

			//----------------------------------------------------------------------------
			// 結果メモリ転送 :出力は「dst_dataとdst_NumBits」の２つ
			//============================================================================
			//result_size = out_bit_stream.status()[YCC_size / 64 - 1]._byte_pos
			//	+ (out_bit_stream.status()[YCC_size / 64 - 1]._bit_pos == 7 ? 0 : 1);
			result_size = out_bit_stream.available_size();
			encode_result.sync_to_host();
		}
		watch.lap();
		watch.stop();
		std::cout << "	" << watch.getLastElapsedTime() << "[ms]\n" << std::endl;
	}

	BitmapCVUtil result(width, height, 8, source.getBytePerPixel());
	{
		//----------------------------------------------------------------------------
		// 色変換テーブルの作成 Decode用定数転送
		//============================================================================
		cuda_memory<int> itrans_table_Y(Y_size);
		cuda_memory<int> itrans_table_C(Y_size);
		make_itrans_table(itrans_table_Y.host_data(), itrans_table_C.host_data(), width, height);
		itrans_table_C.sync_to_device();
		itrans_table_Y.sync_to_device();

		//----------------------------------------------------------------------------
		// メモリ確保
		//============================================================================
		device_memory<byte> src(Y_size * 3);
		device_memory<int> yuv_buffer(YCC_size);
		cuda_memory<int> quantized(YCC_size);
		device_memory<int> dct_coeficient(YCC_size);
		device_memory<float> dct_tmp_buffer(YCC_size);

		std::cout << "	-----------------------------------------------" << std::endl;
		std::cout << "	 Decode" << std::endl;
		std::cout << "	-----------------------------------------------" << std::endl;
		watch.start();
		{
			//----------------------------------------------------------------------------
			// メモリ確保
			//============================================================================
			InBitStream mIBSP(encode_result.host_data(), result_size);

			//----------------------------------------------------------------------------
			// ハフマン復号
			//============================================================================
			// CPU
			decode_huffman(&mIBSP, quantized.host_data(), width, height);
			quantized.sync_to_device();

			//----------------------------------------------------------------------------
			// 逆量子化
			//============================================================================
			gpu_izig_quantize_Y<<<Dg2_0, Db2_0>>>(quantized.device_data(), dct_coeficient.device_data());
			gpu_izig_quantize_C<<<Dg2_1, Db2_1>>>(quantized.device_data(), dct_coeficient.device_data(), Y_size);

			//----------------------------------------------------------------------------
			// 逆DCT
			//============================================================================
			gpu_idct_0<<<Dg1, Db1>>>(dct_coeficient.device_data(), dct_tmp_buffer.device_data());
			gpu_idct_1<<<Dg1, Db1>>>(dct_tmp_buffer.device_data(), yuv_buffer.device_data());

			//----------------------------------------------------------------------------
			// yuv->RGB
			//============================================================================
			gpu_color_itrans<<<Dg0_0, Db0_0>>>(yuv_buffer.device_data(), src.device_data(),
				itrans_table_Y.device_data(), itrans_table_C.device_data(), C_size);

			//----------------------------------------------------------------------------
			// 結果転送
			//============================================================================
			src.copy_host((byte*) result.getRawData(), src.size());
		}
		watch.lap();
		watch.stop();
		std::cout << "	" << watch.getLastElapsedTime() << "[ms]\n" << std::endl;
	}
	result.saveToFile("gpu_" + out_file_name);

	std::cout << "-----------------------------------------------" << std::endl;
	std::cout << " Finish GPU Encoding & Decoding" << std::endl;
	std::cout << "===============================================\n\n" << std::endl;
}

