#include <cstdlib>
#include <cstring>
#include <cmath>

#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>

#include "cpu_jpeg.h"
#include "gpu_jpeg.cuh"

#include "utils/util_cv.h"
#include "utils/timer.h"
#include "utils/cuda_timer.h"
#include "utils/cuda_memory.hpp"
#include "utils/util_cv.h"
#include "utils/encoder_tables.h"
#include "utils/in_bit_stream.h"
#include "utils/gpu_in_bit_stream.cuh"

using namespace std;

static void parse_arg(int argc, char *argv[], string &in_file, string &out_file) {
	if (argc == 3) {
		in_file = argv[1];
		out_file = argv[2];
	} else {
		cout << "Please input source file." << endl;
		abort();
	}
}

void gpu_exec(int argc, char *argv[]) {
	//----------------------------------------------------------------------------
	// 画像読み込み
	//============================================================================
	string file_name, out_file_name;
	parse_arg(argc, argv, file_name, out_file_name);

	BitmapCVUtil source(file_name, BitmapCVUtil::RGB_COLOR);
	const int width = source.getWidth();
	const int height = source.getHeight();
	const int Y_size = width * height;
	const int C_size = Y_size / 4;
	const int YCC_size = Y_size + C_size * 2;

	//----------------------------------------------------------------------------
	// 色変換テーブルの作成
	//============================================================================
	CudaMemory<int> trans_table_Y(width * height);
	CudaMemory<int> trans_table_C(width * height);
	make_trans_table(trans_table_Y.host_data(), trans_table_C.host_data(), width, height);

	CudaMemory<int> itrans_table_Y(width * height);
	CudaMemory<int> itrans_table_C(width * height);
	make_itrans_table(itrans_table_Y.host_data(), itrans_table_C.host_data(), width, height);

	//----------------------------------------------------------------------------
	// ハフマン符号化用メモリ確保
	//============================================================================
	CudaMemory<GPUOutBitStream> out_bit_streams(YCC_size / 64);
	out_bit_streams.syncDeviceMemory();

	ByteBuffer dst_NumBits((width * height + 2 * (width / 2) * (height / 2)) / 64);

	GPUOutBitStreamBufferPointer stream_buffers(sizeof(byte) * (YCC_size / 64) * MBS);

	CudaMemory<byte> encode_result(sizeof(byte) * (Y_size * 3));
	encode_result.fillZero();

	//----------------------------------------------------------------------------
	// Encode用定数転送,コンスタントメモリも使ってみたい
	//============================================================================
	//先に送っておくもの
	trans_table_C.syncDeviceMemory();
	trans_table_Y.syncDeviceMemory();

	//----------------------------------------------------------------------------
	// Decode用定数転送,コンスタントメモリも使ってみたい
	//============================================================================
	itrans_table_C.syncDeviceMemory();
	itrans_table_Y.syncDeviceMemory();

	DeviceMemory<int> yuv_buffer(width * height + 2 * (width / 2) * (height / 2));

	//----------------------------------------------------------------------------
	// Decode用メモリ確保,コンスタントメモリも使ってみたい
	//============================================================================
	DeviceMemory<int> dct_coeficient(width * height + 2 * (width / 2) * (height / 2));
	DeviceMemory<float> dct_tmp_buffer(width * height + 2 * (width / 2) * (height / 2));
	DeviceMemory<int> quantized(width * height + 2 * (width / 2) * (height / 2));
	DeviceMemory<byte> src(width * height * 3);

	//----------------------------------------------------------------------------
	// カーネルDimension設定
	//============================================================================
	const int THREADS = 256;

	const int DCT4_TH = 1;

	const int QUA0_TH = 64;
	const int QUA1_TH = 64;

	const int HUF0_TH = 16;
	const int HUF1_TH = 4; //divide使うなら最速

	dim3 Dg0_0(width * height / THREADS, 1, 1), Db0_0(THREADS, 1, 1);
	dim3 Dg0_1(width * height / THREADS / 2, 1, 1), Db0_1(height / 2, 1, 1);

	dim3 Dg1((width * height + 2 * (width / 2) * (height / 2)) / 64 / DCT4_TH, 1, 1), Db1(DCT4_TH,
		8, 8); //DCT4_THは16が最大

	dim3 Dg2_0(Y_size / QUA0_TH, 1, 1), Db2_0(QUA0_TH, 1, 1);
	dim3 Dg2_1((2 * C_size) / QUA1_TH, 1, 1), Db2_1(QUA1_TH, 1, 1);

	dim3 Dg3_0(YCC_size / 64 / HUF0_TH, 1, 1), Db3_0(HUF0_TH, 1, 1); //YCC_size
	dim3 Dg3_1(YCC_size / 64 / HUF1_TH, 1, 1), Db3_1(HUF1_TH, 1, 1); //YCC_size

	//----------------------------------------------------------------------------
	// ここより前は前処理想定
	//============================================================================
	//----------------------------------------------------------------------------
	// Encode
	//============================================================================
	//----------------------------------------------------------------------------
	// 画像読み込み
	//============================================================================
	BitmapCVUtil result(width, height, 8, source.getBytePerPixel());

	//----------------------------------------------------------------------------
	// メモリ転送
	//============================================================================
	src.write((byte*) source.getRawData(), width * height * 3);

	//----------------------------------------------------------------------------
	// RGB->yuv
	//============================================================================
	gpu_color_trans_Y<<<Dg0_0, Db0_0>>>(src.device_data(), yuv_buffer.device_data(), trans_table_Y.device_data());
	gpu_color_trans_C<<<Dg0_0, Db0_0>>>(src.device_data(), yuv_buffer.device_data(), trans_table_C.device_data(), height, C_size);

	//----------------------------------------------------------------------------
	// DCT
	//============================================================================
	gpu_dct_0<<<Dg1, Db1>>>(yuv_buffer.device_data(), dct_tmp_buffer.device_data());
	gpu_dct_1<<<Dg1, Db1>>>(dct_tmp_buffer.device_data(), dct_coeficient.device_data());

	//----------------------------------------------------------------------------
	// 量子化
	//============================================================================
	gpu_zig_quantize_Y<<<Dg2_0, Db2_0>>>(dct_coeficient.device_data(), quantized.device_data());
	gpu_zig_quantize_C<<<Dg2_1, Db2_1>>>(dct_coeficient.device_data(), quantized.device_data(), Y_size);
	//----------------------------------------------------------------------------
	// ハフマン符号化
	//============================================================================
	gpu_huffman_mcu<<<Dg3_0, Db3_0>>>(quantized.device_data(), out_bit_streams.device_data(), stream_buffers.getWriteBufAddress(), stream_buffers.getEndOfBuf(), width, height);

	// 逐次処理のためCPUに戻す
	out_bit_streams.syncHostMemory();
	cpu_huffman_middle(out_bit_streams.host_data(), width, height, dst_NumBits.data());
	out_bit_streams.syncDeviceMemory();

	gpu_huffman_write_devide0<<<Dg3_1, Db3_1>>>(out_bit_streams.device_data(), stream_buffers.getWriteBufAddress(), encode_result.device_data(), width, height);
	gpu_huffman_write_devide1<<<Dg3_1, Db3_1>>>(out_bit_streams.device_data(), stream_buffers.getWriteBufAddress(), encode_result.device_data(), width, height);
	gpu_huffman_write_devide2<<<Dg3_1, Db3_1>>>(out_bit_streams.device_data(), stream_buffers.getWriteBufAddress(), encode_result.device_data(), width, height);

	//----------------------------------------------------------------------------
	// 結果メモリ転送 :出力は「dst_dataとdst_NumBits」の２つ
	//============================================================================
	int dst_size = out_bit_streams[YCC_size / 64 - 1].mBytePos
		+ (out_bit_streams[YCC_size / 64 - 1].mBitPos == 7 ? 0 : 1);
	encode_result.syncHostMemory();

	//----------------------------------------------------------------------------
	// Decode
	//============================================================================
	//----------------------------------------------------------------------------
	// メモリ確保
	//============================================================================
	InBitStream mIBSP(encode_result.host_data(), dst_size);
	IntBuffer c_qua(width * height + 2 * (width / 2) * (height / 2));

	//----------------------------------------------------------------------------
	// ハフマン復号
	//============================================================================
	// CPU
	decode_huffman(&mIBSP, c_qua.data(), width, height);
	// GPU:GPUInstream.hにバグがある可能性もあるので留意
	hipMemcpy(quantized.device_data(), c_qua.data(),
		sizeof(int) * (width * height + 2 * (width / 2) * (height / 2)), hipMemcpyHostToDevice);

	//----------------------------------------------------------------------------
	// 逆量子化
	//============================================================================
	gpu_izig_quantize_Y<<<Dg2_0, Db2_0>>>(quantized.device_data(), dct_coeficient.device_data());
	gpu_izig_quantize_C<<<Dg2_1, Db2_1>>>(quantized.device_data(), dct_coeficient.device_data(), Y_size);

	//----------------------------------------------------------------------------
	// 逆DCT
	//============================================================================
	gpu_idct_0<<<Dg1, Db1>>>(dct_coeficient.device_data(), dct_tmp_buffer.device_data());
	gpu_idct_1<<<Dg1, Db1>>>(dct_tmp_buffer.device_data(), yuv_buffer.device_data());

	//----------------------------------------------------------------------------
	// yuv->RGB
	//============================================================================
	gpu_color_itrans<<<Dg0_0, Db0_0>>>(yuv_buffer.device_data(), src.device_data(), itrans_table_Y.device_data(), itrans_table_C.device_data(), C_size);

	//----------------------------------------------------------------------------
	// 結果転送
	//============================================================================
	hipMemcpy((byte*) result.getRawData(), src.device_data(), src.size(), hipMemcpyDeviceToHost);

	out_file_name = "gpu_" + out_file_name;
	result.saveToFile(out_file_name);
}

