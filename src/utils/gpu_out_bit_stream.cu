#include "hip/hip_runtime.h"
﻿/******************************************************
 COutBitStream Ver.1.1.0 ビットストリーム・クラス
 Copyright 2004 AsWe.Co.Ltd. All rights reserved.
 # The comments are written in Japanese (Shift-JIS).
 ******************************************************/

#include <hip/hip_runtime.h>
#include <cstring>

#include "gpu_out_bit_stream.cuh"


#ifdef __CDT_PARSER__
#define __global__
#define __device__
#define __host__
#endif

GPUOutBitStream::GPUOutBitStream() :
	mBytePos(0),
	mBitPos(7), // 最上位ビット
	mWriteFlag(1), // アクセス許可
	mNumBits(0) {
}

//void InitGPUBuffer(GPUOutBitStreamBufferPointer* bp, int size) {
//	hipMemset(bp->HeadOfBuf, 0, sizeof(byte) * size);
//	bp->WriteBufAddress = bp->HeadOfBuf;
//	bp->EndOfBuf = bp->HeadOfBuf + size; // バッファの最終アドレス
//}

