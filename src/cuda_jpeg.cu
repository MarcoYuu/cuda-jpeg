#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "cuda_jpeg.cuh"

namespace jpeg {
	namespace cuda {

		using namespace util;

#ifdef DEBUG
		__global__ void ConvertRGBToYUV(
			const byte* rgb,
			byte* yuv_result,
			size_t width,
			size_t height,
			size_t block_width,
			size_t block_height, int *result) {
#else
		__global__ void ConvertRGBToYUV(
			const byte* rgb,
			byte* yuv_result,
			size_t width,
			size_t height,
			size_t block_width,
			size_t block_height) {
#endif
			// ------------------- 各CUDAブロックに対して
			const int grid_row_num = height / block_height;
			const int grid_col_num = width / block_width;
			const int grid_x = blockIdx.z / grid_col_num;
			const int grid_y = blockIdx.z % grid_col_num;

			// 元画像の各画像ブロックに対する左上インデックス
			const int src_start_index = grid_y * block_width * 3 + grid_x * width * block_height * 3;

			// 書き込み先のブロックごとの先頭アドレス
			const int dst_start_index = blockIdx.z * block_width * block_height * 3 / 2;

			// ------------------- CUDAブロック内の任意画像ブロック分割に対して
			// 画像ブロック内の16x16ブロックの左上アドレス
			const int src_block_start_index = src_start_index + blockIdx.x * 16 * 3 + blockIdx.y * width * 3 * 16;

			// 書き込み先の16x16ブロックごとの先頭アドレス
			const int dst_block_start_y_index = dst_start_index + blockIdx.x * 16 * 16
				+ blockIdx.y * 16 * 16 * block_width / 16;
			const int dst_block_start_u_index = dst_block_start_y_index + block_width * block_height / 4;
			const int dst_block_start_v_index = dst_block_start_u_index + block_width * block_height / 4;

			// ------------------- 16x16 block に関して
			const int x = threadIdx.x;
			const int y = threadIdx.y;
			const int mcu_id_x = x / 8;
			const int mcu_id_y = y / 8;
			const int mcu_id = mcu_id_x + mcu_id_y * 2; // 0-4
			const int mcu_offset = mcu_id * 64; // 0, 64, 128, 192

			// 元画像の8x8ごとのインデックス
			const int src_8x8_left_up = src_block_start_index + mcu_id_x * 8 * 3 + mcu_id_y * width * 3 * 8;
			const int local_index = x % 8 * 3 + (y % 8) * width * 3;
			const int src_id = mcu_offset + local_index;

			// 書き込み先インデックス
			const int local_dst_index = x % 8 + (y % 8) * 8; // 0-63
			const int dst_id = dst_block_start_y_index + mcu_offset + local_dst_index;

#ifdef DEBUG
			result[dst_id] = src_id;
#endif

			// 色変換
			//Y [0,255]
			yuv_result[dst_id] = byte(
				0.1440 * rgb[src_id] + 0.5870 * rgb[src_id + 1] + 0.29990 * rgb[src_id + 2]);

			printf("%d, %d\n", src_id, rgb[src_id + 2]);

			//U,V [-128,127] -> [0,255]
			if (x % 2 == 0 && y % 2 == 0) {
				const int local_dst_c_index = x / 2 + y / 8 * 8; // 0-63
				const int dst_u_id = dst_block_start_u_index + local_dst_c_index;
				const int dst_v_id = dst_block_start_v_index + local_dst_c_index;
				yuv_result[dst_u_id] = byte(
					0.5000 * rgb[src_id] - 0.3313 * rgb[src_id + 1] - 0.1687 * rgb[src_id + 2] + 128);
				yuv_result[dst_v_id] = byte(
					-0.0813 * rgb[src_id] - 0.4187 * rgb[src_id + 1] + 0.5000 * rgb[src_id + 2] + 128);
			}
		}

		__global__ void ConvertYUVToRGB(const byte* yuv, byte* rgb_result, size_t width, size_t height,
			size_t block_width, size_t block_height) {

			// ------------------- 各CUDAブロックに対して
			const int grid_col_num = width / block_width;
			const int grid_x = blockIdx.z / grid_col_num;
			const int grid_y = blockIdx.z % grid_col_num;

			const int block_byte_size = block_width * block_height * 3 / 2;

			// 元画像の各画像ブロックに対する左上インデックス
			const int src_start_index = grid_x * block_byte_size + grid_y * block_byte_size * grid_col_num;

			// ------------------- CUDAブロック内の任意画像ブロック分割に対して
			// 画像ブロック内の16x16ブロックの左上アドレス
			const int src_block_start_y_index = src_start_index + blockIdx.x * 256 + blockIdx.y * blockDim.x * 256;
			const int src_block_start_u_index = src_block_start_y_index + block_width * block_height / 4
				+ blockIdx.x * 64
				+ blockIdx.y * blockDim.x * 64;
			const int src_block_start_v_index = src_block_start_y_index + block_width * block_height / 2
				+ blockIdx.x * 64
				+ blockIdx.y * blockDim.x * 64;

			// ------------------- 16x16 block に関して
			const int x = threadIdx.x;
			const int y = threadIdx.y;
			const int mcu_id_x = x / 8; // 0,1
			const int mcu_id_y = y / 8; // 0,1
			const int mcu_id = mcu_id_x + mcu_id_y * 2; // 0-4
			const int mcu_offset = mcu_id * 64; // 0, 64, 128, 192
			const int local_src_index = x % 8 + (y % 8) * 8; // 0-63
			const int src_id = src_block_start_y_index + local_src_index;
		}
	}
}
